#include "hip/hip_runtime.h"
//
//  Piano Video
//  A free piano visualizer.
//  Copyright Patrick Huang 2021
//
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <https://www.gnu.org/licenses/>.
//

/*
Cuda implementations for util functions.
*/

#include "utils_cu.cuh"


__host__ __device__ double radians(CD deg) {
    /*
    Convert degrees to radians.
    */
    return deg / 180 * PI;
}

__host__ __device__ double degrees(CD rad) {
    /*
    Radians to degrees.
    */
    return rad / PI * 180;
}

__host__ __device__ double pythag(CD dx, CD dy) {
    /*
    Pythagorean distance.

    :param dx: X delta.
    :param dy: Y delta.
    */
    return std::pow((dx*dx) + (dy*dy), 0.5);
}

__host__ __device__ int ibounds(const int v, const int vmin, const int vmax) {
    /*
    Integer bounds.

    :param v: Value.
    :param vmin: Minimum value.
    :param vmax: Maximum value.
    */
    return min(max(v, vmin), vmax);
}

__host__ __device__ double dbounds(CD v, CD vmin, CD vmax) {
    /*
    Double bounds.

    :param v: Value.
    :param vmin: Minimum value.
    :param vmax: Maximum value.
    */
    return min(max(v, vmin), vmax);
}

__host__ __device__ double map_range(CD v, CD old_min, CD old_max, CD new_min, CD new_max) {
    CD fac = (v-old_min) / (old_max-old_min);
    CD mapped = fac * (new_max-new_min) + new_min;
    return mapped;
}


__host__ __device__ bool is_white(const UCH key) {
    const UCH num = (key-3) % 12;
    switch (num) {
        case 1: return false;
        case 3: return false;
        case 6: return false;
        case 8: return false;
        case 10: return false;
        default: return true;
    }
}

__host__ __device__ double key_pos(CD start, CD end, const UCH key) {
    // FIXME BUGGY
    CD white_width = (end-start) / 52.0;

    double x = 0;
    for (UCH i = 0; i < key; i++) {
        if (is_white(i))
            x += white_width;
    }
    if (is_white(key))
        x += white_width/2.0;

    return x;
}
